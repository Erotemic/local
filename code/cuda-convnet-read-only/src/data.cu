#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <algorithm>
#include <data.cuh>

using namespace std;

DataProvider::DataProvider(int minibatchSize) : 
    _minibatchSize(minibatchSize), _hData(NULL) {

}

GPUData& DataProvider::operator[](int idx) {
    return getMinibatch(idx);
}

void DataProvider::clearData() {
    delete _hData;
    _hData = NULL;
    _dataSize = 0;
}

void DataProvider::setData(CPUData& hData) {
    // This is now deleted by the DataWorker's destructor
//    delete _hData; // Delete old CPU matrices

    _hData = &hData;
    _dataSize = 0;
    for (int i = 0; i < hData.getSize(); i++) {
        _dataSize += hData[i].getNumDataBytes();
    }
    _dataSize /= 1024 * 1024;
    if (_dataSize < MAX_DATA_ON_GPU) {
        for (int i = 0; i < hData.getSize(); i++) {
            if (i >= _data.size()) {
                _data.push_back(new NVMatrix());
            }
            _data[i]->copyFromHost(hData[i], true);
        }
    }
}

GPUData& DataProvider::getMinibatch(int idx) {
    assert(idx >= 0 && idx < getNumMinibatches());
    return getDataSlice(idx * _minibatchSize, (idx + 1) * _minibatchSize);
}

GPUData& DataProvider::getDataSlice(int startCase, int endCase) {
    assert(_hData != NULL);
    assert(_hData->getNumCases() > 0);
    
    NVMatrixV& miniData = *new NVMatrixV();
    
    for (int i = 0; i < _hData->getData().size(); i++) {
        miniData.push_back(new NVMatrix());
        if (_dataSize < MAX_DATA_ON_GPU) {
            if (_data[i]->isTrans()) {
                _data[i]->sliceRows(startCase, min(_hData->getNumCases(), endCase), *miniData[i]);
            } else {
                _data[i]->sliceCols(startCase, min(_hData->getNumCases(), endCase), *miniData[i]);
            }
        } else {
            Matrix tmp;
            if ((*_hData)[i].isTrans()) {
                (*_hData)[i].sliceRows(startCase, min(_hData->getNumCases(), endCase), tmp);
            } else {
                (*_hData)[i].sliceCols(startCase, min(_hData->getNumCases(), endCase), tmp);
            }
            miniData.back()->copyFromHost(tmp, true);
        }
    }

    return *new GPUData(miniData);
}

int DataProvider::getNumMinibatches() {
    assert(_hData != NULL);
    assert(_hData->getNumCases() > 0);
    return DIVUP(_hData->getNumCases(), _minibatchSize);
}

int DataProvider::getMinibatchSize() {
    return _minibatchSize;
}

int DataProvider::getNumCases() {
    assert(_hData != NULL);
    assert(_hData->getNumCases() > 0);
    return _hData->getNumCases();
}

int DataProvider::getNumCasesInMinibatch(int idx) {
    assert(_hData != NULL);
    assert(_hData->getNumCases() > 0);
    assert(idx >= 0 && idx < getNumMinibatches());
    return min(_minibatchSize, max(0, _hData->getNumCases() - idx * _minibatchSize));
}